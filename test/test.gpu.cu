#include "hip/hip_runtime.h"
#include <cassert>
#include <stdio.h>

#include "Manager.h"
#include "node_dsl.hpp"


#define CATCH_CONFIG_MAIN
#include <catch/catch.hpp>


#define COLUMNS 3
#define ROWS 2

__global__ void add(int *a, int *b, int *c)
{
	int x = blockIdx.x;
	int y = blockIdx.y;
	int i = (COLUMNS*y) + x;
	c[i] = a[i] + b[i];
}

void cuda_func() {
	int a[ROWS][COLUMNS], b[ROWS][COLUMNS], c[ROWS][COLUMNS];
	int *dev_a, *dev_b, *dev_c;
	
	hipMalloc((void **) &dev_a, ROWS*COLUMNS*sizeof(int));
	hipMalloc((void **) &dev_b, ROWS*COLUMNS*sizeof(int));
	hipMalloc((void **) &dev_c, ROWS*COLUMNS*sizeof(int));
	
	for (int y = 0; y < ROWS; y++){// Fill Arrays
		for (int x = 0; x < COLUMNS; x++)
		{
			a[y][x] = x;
			b[y][x] = y;
		}
	}

	hipMemcpy(dev_a, a, ROWS*COLUMNS*sizeof(int),
	hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, ROWS*COLUMNS*sizeof(int),
	hipMemcpyHostToDevice);
	
	dim3 grid(COLUMNS,ROWS);
	add<<<grid,1>>>(dev_a, dev_b, dev_c);
	
	hipMemcpy(c, dev_c, ROWS*COLUMNS*sizeof(int),
	hipMemcpyDeviceToHost);
}

TEST_CASE( "CUDA Graph execution order is correct.", "[CUDA-manager]" ) {
	auto fun0 = []() {};
	auto fun1 = []() { std::this_thread::sleep_for(std::chrono::microseconds(500)); };

	Manager m;

	auto& node0 = m.append_node(0, fun0);
	auto& node1 = m.append_node(1, fun1);
	auto& node2 = m.append_node(2, fun0);
	auto& node3 = m.append_node(3, fun0);
	auto& node4 = m.append_node(4, cuda_func);

	node0 >> (node1, node2) >> node3 >> node4;
	m.execute(0);

	std::vector<int> expected_order = {0, 2, 1, 3, 4};
	if (std::thread::hardware_concurrency() == 1){
		expected_order = {0, 1, 2, 3, 4};
	}
	REQUIRE(m.execution_order() == expected_order);
}
