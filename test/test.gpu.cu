#include <cassert>
#include <stdio.h>

#include "Manager.h"
#include "node_dsl.hpp"
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>

#define CATCH_CONFIG_MAIN
#include <catch/catch.hpp>

int cuda_func()
{
  // allocate three device_vectors with 10 elements
  thrust::device_vector<int> X(10);
  thrust::device_vector<int> Y(10);
  thrust::device_vector<int> Z(10);

  // initialize X to 0,1,2,3, ....
  thrust::sequence(X.begin(), X.end());

  // compute Y = -X
  thrust::transform(X.begin(), X.end(), Y.begin(), thrust::negate<int>());

  // fill Z with twos
  thrust::fill(Z.begin(), Z.end(), 2);

  // compute Y = X mod 2
  thrust::transform(X.begin(), X.end(), Z.begin(), Y.begin(), thrust::modulus<int>());

  // replace all the ones in Y with tens
  thrust::replace(Y.begin(), Y.end(), 1, 10);
 
  return 0; 
}

TEST_CASE( "CUDA Graph execution order is correct.", "[CUDA-manager]" ) {
	auto fun0 = []() {};
	auto fun1 = []() { std::this_thread::sleep_for(std::chrono::microseconds(5000)); };

	Manager m;

	auto& node0 = m.append_node(0, fun0);
	auto& node1 = m.append_node(1, fun1);
	auto& node2 = m.append_node(2, fun0);
	auto& node3 = m.append_node(3, fun0);
	auto& node4 = m.append_node(4, cuda_func);

	node0 >> (node1, node2) >> node3 >> node4;
	m.execute(0);

	std::vector<int> expected_order = {0, 2, 1, 3, 4};
	if (std::thread::hardware_concurrency() == 1){
		expected_order = {0, 1, 2, 3, 4};
	}
	REQUIRE(m.execution_order() == expected_order);
}
